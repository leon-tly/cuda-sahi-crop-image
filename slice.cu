#include "hip/hip_runtime.h"
#include "slice.h"

__global__ void slice_kernel(
  const uint8_t*  image,
  uint8_t*  outs,
  const int width,
  const int height,
  const int slice_width,
  const int slice_height,
  const int slice_num_h,
  const int slice_num_v,
  const int overlap_pixel)
{
    const int out_size = 3 * slice_width * slice_height;
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if (dx >= width || dy >= height)
    {
        return;
    }
    int offset = dy * width + dx;
    uint8_t b = image[3 * offset + 0];
    uint8_t g = image[3 * offset + 1];
    uint8_t r = image[3 * offset + 2];
    for (int i = 0; i < slice_num_h; i++)
    {
        int sdx_start = MAX(0, i * slice_width - overlap_pixel);
        int sdx_end   = sdx_start + slice_width;
        for (int j = 0; j < slice_num_v; j++)
        {
            int sdy_start = MAX(0, j * slice_height - overlap_pixel);
            int sdy_end   = sdy_start + slice_height;
            if (dx >= sdx_start && dx < sdx_end && dy >= sdy_start && dy < sdy_end)
            {
                int image_id = i * slice_num_h + j;
                int sdx = dx - sdx_start;
                int sdy = dy - sdy_start;
                int soffset = sdy * slice_width + sdx;
                outs[image_id * out_size + 3 * soffset + 0] = b;
                outs[image_id * out_size + 3 * soffset + 1] = g;
                outs[image_id * out_size + 3 * soffset + 2] = r;
            }
        }
    }
}

void slice(
  const uint8_t* data,
  std::vector<cv::Mat>& slice_images, 
  std::vector<cv::Rect_<float>>& crop_size,
  const int width,
  const int height,
  const int slice_num_h, 
  const int slice_num_v, 
  const float overlap_ratio)
{
    int slice_num = slice_num_h * slice_num_v;
    const int overlap_pixel = std::max(width, height) * overlap_ratio;
    const int slice_width = (width - overlap_pixel) / slice_num_h + overlap_pixel;
    const int slice_height = (height - overlap_pixel) / slice_num_v + overlap_pixel;
    int output_img_size = 3 * slice_width * slice_height;
    uint8_t* output_imgs_gpu;
    hipMalloc((void**)&output_imgs_gpu, output_img_size * slice_num * sizeof(uint8_t));
    hipMemset(output_imgs_gpu, 114, output_img_size * slice_num * sizeof(uint8_t));
    dim3 threads(32, 32);
    dim3 blocks((width + 31) / 32, (height + 31) / 32);

    slice_kernel<<<blocks, threads>>>(data, output_imgs_gpu, 
                                            width, height, 
                                            slice_width, slice_height, 
                                            slice_num_h, slice_num_v, overlap_pixel);
    hipDeviceSynchronize();
    slice_images.resize(slice_num);
    crop_size.resize(slice_num);
    for (int i = 0; i < slice_num_h; i++)
    {
        int x = MAX(0, i * slice_width - overlap_pixel);
        for (int j = 0; j < slice_num_v; j++)
        {
            int y = MAX(0, j * slice_height - overlap_pixel);
            int image_id = i * slice_num_h + j;
            crop_size[image_id] = cv::Rect_<float>(cv::Point_<float>(x, y), cv::Point_<float>(x+slice_width, y+slice_height));
            slice_images[image_id] = cv::Mat(slice_height, slice_width, CV_8UC3);
            uint8_t* output_img_data = slice_images[image_id].ptr<uint8_t>();
            hipMemcpy(output_img_data, output_imgs_gpu+image_id*output_img_size, output_img_size*sizeof(uint8_t), hipMemcpyDeviceToHost);
        }
    }
    hipFree(output_imgs_gpu);
}
